#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>
#ifdef _OPENMP
#include <omp.h>
#endif

#define TILE_DIM     24
#define THREAD_COUNT 512
#define STRIP_HEIGHT 16

// void print_shape(torch::Tensor data)
// {
//     auto num_dims = data.dim();
//     std::cout << "Shape: (";
//     for (int dim = 0; dim < num_dims; dim++)
//     {
//         std::cout << data.size(dim);
//         if (dim != num_dims - 1)
//         {
//             std::cout << ", ";
//         }
//         else
//         {
//             std::cout << ")" << std::endl;
//         }
//     }
// }

__device__ float l1distance_cuda(const float &in1, const float &in2)
{
    return abs(in1 - in2);
}


float l1distance_cuda(const float *in1, const float *in2, int size)
{
    float ret_sum = 0.0;
    for (int c_i = 0; c_i < size; c_i++)
    {
        ret_sum += abs(in1[c_i] - in2[c_i]);
    }
    return ret_sum;
}



// float l2distance(const float *in1, const float *in2, int size)
// {
//     float ret_sum = 0.0;
//     for (int c_i = 0; c_i < size; c_i++)
//     {
//         ret_sum += (in1[c_i] - in2[c_i]) * (in1[c_i] - in2[c_i]);
//     }
//     return std::sqrt(ret_sum);
// }

template <typename scalar_t>
__global__ void geodesic_updown_pass_kernel(
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> image_ptr, 
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> distance_ptr,
    const float &l_grad, 
    const float &l_eucl,
    const float *local_dist,
    const int row,
    const int height,
    const int width)
{
    __shared__ float lastRow[THREAD_COUNT+2];
    __shared__ float curRow[THREAD_COUNT+2];

    // overlap
    int blockStartX = (blockIdx.x * THREAD_COUNT) - (blockIdx.x * STRIP_HEIGHT * 2);
    int blockSafeX = blockStartX + THREAD_COUNT - (2 * STRIP_HEIGHT);

    int kernelX = blockStartX + threadIdx.x;

    // copy last and current row into shared memory using each thread
    lastRow[threadIdx.x + 1] = -1;// distance_ptr[0][0][row-1][kernelX];
    curRow[threadIdx.x + 1] = -1;// distance_ptr[0][0][row][kernelX];
    if(kernelX < width)
    {
        lastRow[threadIdx.x + 1] = distance_ptr[0][0][row-1][kernelX];
        curRow[threadIdx.x + 1] = distance_ptr[0][0][row][kernelX];
    }
    
    if (threadIdx.x == 0)
    {
        lastRow[0] = -1;
        curRow[0] = -1;
        if ((kernelX - 1) >= 0 && (kernelX-1) < width)
        { 
            lastRow[0] = distance_ptr[0][0][row-1][kernelX-1];
            curRow[0] = distance_ptr[0][0][row][kernelX-1];
        }
    }
    else if(threadIdx.x == THREAD_COUNT + 1)
    {
        lastRow[threadIdx.x + 1] = -1;
        curRow[threadIdx.x + 1] = -1;
        if ((kernelX+1) >= 0 && (kernelX + 1) < width)
        {
            lastRow[threadIdx.x + 1] = distance_ptr[0][0][row-1][kernelX+1];
            curRow[threadIdx.x + 1] = distance_ptr[0][0][row][kernelX+1];
        }
    }

    __syncthreads();

    // top-down pass for each row in strip
    for (int i = 0; i < STRIP_HEIGHT; i++)
    {
        float solution = -1;
        int currentHeightIdx = row + i;

        if (currentHeightIdx < height and kernelX < width)
        {
            int localKernelX = (int)threadIdx.x + 1;
            // solution = curRow[localKernelX];

            float pval = image_ptr[0][0][currentHeightIdx][kernelX];
            
            int w_i, w_ind;
            float cur_dist;

            // left back
            w_i = 0;
            w_ind = kernelX + w_i - 1;
            cur_dist = lastRow[localKernelX + w_i - 1];
            float left_solution=-1;
            if (cur_dist >= 0 && (w_ind >= 0 && w_ind < width))
            {
                float qval = image_ptr[0][0][currentHeightIdx-1][w_ind];
                float l_dist = abs(pval - qval);
                left_solution = (cur_dist + l_eucl * local_dist[w_i] + l_grad * l_dist);
            }

            // center back
            w_i = 1;
            w_ind = kernelX + w_i - 1;
            cur_dist = lastRow[localKernelX + w_i - 1];
            float center_solution=-1;
            if (cur_dist >= 0 && (w_ind >= 0 && w_ind < width))
            {
                float qval = image_ptr[0][0][currentHeightIdx-1][w_ind];
                float l_dist = abs(pval - qval);
                center_solution = (cur_dist + l_eucl * local_dist[w_i] + l_grad * l_dist);
            }

            // right back
            w_i = 2;
            w_ind = kernelX + w_i - 1;
            cur_dist = lastRow[localKernelX + w_i - 1];
            float right_solution=-1;
            if (cur_dist >= 0 && (w_ind >= 0 && w_ind < width))
            {
                float qval = image_ptr[0][0][currentHeightIdx-1][w_ind];
                float l_dist = abs(pval - qval);
                right_solution = (cur_dist + l_eucl * local_dist[w_i] + l_grad * l_dist);
            }

            // for(int w_i = 0; w_i < 3; w_i++)
            // {
            //     const int w_ind = kernelX + w_i - 1;
            //     const float cur_dist = lastRow[localKernelX + w_i - 1];
            //     if (cur_dist >= 0 && (w_ind >= 0 && w_ind < width))
            //     {
            //         float qval = image_ptr[0][0][currentHeightIdx-1][w_ind];
            //         float l_dist = l1distance_cuda(pval, qval);
            //         float cur_solution = (cur_dist + l_eucl * local_dist[w_i] + l_grad * l_dist);
            //         if(w_i == 0)
            //         {
            //             solution = cur_solution;
            //         }
            //         else
            //         {
            //             if(cur_solution < solution)
            //             {
            //                 solution = cur_solution;
            //             }
            //         }
            //    }
            // }
            solution = left_solution;
            if(center_solution < left_solution)
            {
                solution = center_solution;
            }
            else if(right_solution < left_solution && right_solution < center_solution)
            {
                solution = right_solution;
            }
        }
        printf("%d ",kernelX);
        
        __syncthreads();
        
        // if(currentHeightIdx >= 0 && currentHeightIdx < height && kernelX >= 0 && kernelX < width && kernelX < blockSafeX-2)
        // {
        //     distance_ptr[0][0][currentHeightIdx][kernelX] = solution;
        //     // curRow[threadIdx.x + 1] = solution;

        // }
        // if (solution >= 0.0 && kernelX < blockSafeX && (curRow[threadIdx.x] < 0.0 || solution < curRow[threadIdx.x]))
        if (solution >= 0.0 && kernelX < blockSafeX-10 &&  solution < curRow[threadIdx.x + 1])
        {
            printf("%d ",kernelX);
            distance_ptr[0][0][currentHeightIdx][kernelX] = solution;
            curRow[threadIdx.x + 1] = solution;
        }

        lastRow[threadIdx.x] = curRow[threadIdx.x];
        curRow[threadIdx.x] = -1;
        if((currentHeightIdx+1) < height)
        {
            curRow[threadIdx.x] = distance_ptr[0][0][currentHeightIdx + 1][kernelX];     
            if (threadIdx.x == 0)
            {
                curRow[0] = -1;
                if((kernelX - 1) >= 0)
                {
                    curRow[0] = distance_ptr[0][0][currentHeightIdx + 1][kernelX - 1];
                }
            }
            else if (threadIdx.x == THREAD_COUNT + 1)
            {
                curRow[THREAD_COUNT + 1] = -1;
                if((kernelX + 1) < width)
                {
                    curRow[THREAD_COUNT + 1] = distance_ptr[0][0][currentHeightIdx + 1][kernelX + 1];
                }
            }
        }

        __syncthreads();
            
            // TODO: bottom up pass
    }
    

}

void geodesic_updown_pass_cuda(const torch::Tensor &image, torch::Tensor &distance, const float &l_grad,  const float &l_eucl)
{
    // batch, channel, height, width
    const int channel = image.size(1);
    const int height = image.size(2);
    const int width = image.size(3);

    // constexpr float local_dist[] = {sqrt(2.), 1., sqrt(2.)};
    const float local_dist[] = {sqrt(float(2.)), float(1.), sqrt(float(2.))};

    int blockSafeZone = (THREAD_COUNT - (STRIP_HEIGHT * 2));
	int blockCountUpDown = width / blockSafeZone;
	int blockCountLeftRight = height / blockSafeZone;

	if (width % blockSafeZone != 0)
		blockCountUpDown++;
	if (height % blockSafeZone != 0)
		blockCountLeftRight++;

    // process each strip
    for (int row = 1; row < height; row += STRIP_HEIGHT)
    {
        // call kernel
        AT_DISPATCH_FLOATING_TYPES(image.type(), "geodesic_updown_pass_kernel_call", ([&] {
            geodesic_updown_pass_kernel<scalar_t><<<blockCountUpDown, THREAD_COUNT>>>(
                image.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(), 
                distance.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(), 
                l_grad, 
                l_eucl, 
                local_dist, 
                row, 
                height, 
                width);
            }));
    }
}

torch::Tensor generalised_geodesic2d_cuda(torch::Tensor &image, const torch::Tensor &mask, const float &v, const float &l_grad, const float &l_eucl, const int &iterations)
{
    torch::Tensor distance = v * mask.clone();
    image = image.contiguous();
    distance = distance.contiguous();

    std::cout << "Reached here2" << std::endl;

    // top-bottom - width*, height
    geodesic_updown_pass_cuda(image, distance, l_grad, l_eucl);

    // // iteratively run the distance transform
    // for (int itr = 0; itr < iterations; itr++)
    // {
    //     image = image.contiguous();
    //     distance = distance.contiguous();

    //     std::cout << "Reached here2" << std::endl;

    //     // top-bottom - width*, height
    //     geodesic_updown_pass_cuda(image, distance, l_grad, l_eucl);

    //     // // left-right - height*, width
    //     // image = image.transpose(2, 3);
    //     // distance = distance.transpose(2, 3);

    //     // image = image.contiguous();
    //     // distance = distance.contiguous();
    //     // geodesic_updown_pass_cuda(image, distance, l_grad, l_eucl);
        
    //     // // tranpose back to original - width, height
    //     // image = image.transpose(2, 3);
    //     // distance = distance.transpose(2, 3);

    //     // * indicates the current direction of pass
    // }

    return distance;
}

void geodesic_frontback_pass_cuda(const torch::Tensor &image, torch::Tensor &distance, const std::vector<float> &spacing, const float &l_grad, const float &l_eucl)
{
    // batch, channel, depth, height, width
    const int channel = image.size(1);
    const int depth = image.size(2);
    const int height = image.size(3);
    const int width = image.size(4);

    auto image_ptr = image.accessor<float, 5>();
    auto distance_ptr = distance.accessor<float, 5>();

    float local_dist[3*3];
    for (int h_i = 0; h_i < 3; h_i++)
    {
        for (int w_i = 0; w_i < 3; w_i++)
        {
            float ld = spacing[0];
            ld += float(std::abs(h_i-1)) * spacing[1];
            ld += float(std::abs(w_i-1)) * spacing[2];

            local_dist[h_i * 3 + w_i] = ld;
        }
    }

    // front-back
    for (int z = 1; z < depth; z++)
    {
        // use openmp to parallelise the loops over height and width
        #ifdef _OPENMP
            #pragma omp parallel for collapse(2)
        #endif
        for (int h = 0; h < height; h++)
        {
            for (int w = 0; w < width; w++)
            {
                float pval;
                float pval_v[channel];
                if (channel == 1)
                {
                    pval = image_ptr[0][0][z][h][w];
                }
                else
                {
                    for (int c_i = 0; c_i < channel; c_i++)
                    {
                        pval_v[c_i] = image_ptr[0][c_i][z][h][w];
                    }
                }
                float new_dist = distance_ptr[0][0][z][h][w];

                for (int h_i = 0; h_i < 3; h_i++)
                {
                    for (int w_i = 0; w_i < 3; w_i++)
                    {
                        const int h_ind = h + h_i - 1;
                        const int w_ind = w + w_i - 1;

                        if (w_ind < 0 || w_ind >= width || h_ind < 0 || h_ind >= height)
                            continue;

                        float l_dist;
                        if (channel == 1)
                        {
                            l_dist = std::abs(pval - image_ptr[0][0][z - 1][h_ind][w_ind]);
                        }
                        else
                        {
                            float qval_v[channel];
                            for (int c_i = 0; c_i < channel; c_i++)
                            {
                                qval_v[c_i] = image_ptr[0][c_i][z - 1][h_ind][w_ind];
                            }
                            l_dist = l1distance_cuda(pval_v, qval_v, channel);
                        }
                        const float cur_dist = distance_ptr[0][0][z - 1][h_ind][w_ind] + l_eucl * local_dist[h_i * 3 + w_i]  + l_grad * l_dist;
                        new_dist = std::min(new_dist, cur_dist);
                    }
                }
                distance_ptr[0][0][z][h][w] = new_dist;
            }
        }
    }

    // back-front
    for (int z = depth - 2; z >= 0; z--)
    {
        // use openmp to parallelise the loops over height and width
        #ifdef _OPENMP
            #pragma omp parallel for collapse(2)
        #endif
        for (int h = 0; h < height; h++)
        {
            for (int w = 0; w < width; w++)
            {
                float pval;
                float pval_v[channel];
                if (channel == 1)
                {
                    pval = image_ptr[0][0][z][h][w];
                }
                else
                {
                    for (int c_i = 0; c_i < channel; c_i++)
                    {
                        pval_v[c_i] = image_ptr[0][c_i][z][h][w];
                    }
                }
                float new_dist = distance_ptr[0][0][z][h][w];

                for (int h_i = 0; h_i < 3; h_i++)
                {
                    for (int w_i = 0; w_i < 3; w_i++)
                    {
                        const int h_ind = h + h_i - 1;
                        const int w_ind = w + w_i - 1;

                        if (w_ind < 0 || w_ind >= width || h_ind < 0 || h_ind >= height)
                            continue;

                        float l_dist;
                        if (channel == 1)
                        {
                            l_dist = std::abs(pval - image_ptr[0][0][z + 1][h_ind][w_ind]);
                        }
                        else
                        {
                            float qval_v[channel];
                            for (int c_i = 0; c_i < channel; c_i++)
                            {
                                qval_v[c_i] = image_ptr[0][c_i][z + 1][h_ind][w_ind];
                            }
                            l_dist = l1distance_cuda(pval_v, qval_v, channel);
                        }
                        const float cur_dist = distance_ptr[0][0][z + 1][h_ind][w_ind] + l_eucl * local_dist[h_i * 3 + w_i] + l_grad * l_dist;
                        new_dist = std::min(new_dist, cur_dist);
                    }
                }
                distance_ptr[0][0][z][h][w] = new_dist;
            }
        }
    }
}

torch::Tensor generalised_geodesic3d_cuda(torch::Tensor &image, const torch::Tensor &mask, const std::vector<float> &spacing, const float &v, const float &l_grad, const float &l_eucl, const int &iterations)
{
    torch::Tensor distance = v * mask.clone();

    // iteratively run the distance transform
    for (int itr = 0; itr < iterations; itr++)
    {
        image = image.contiguous();
        distance = distance.contiguous();

        // front-back - depth*, height, width
        geodesic_frontback_pass_cuda(image, distance, spacing, l_grad, l_eucl);

        // top-bottom - height*, depth, width
        image = torch::transpose(image, 3, 2);
        distance = torch::transpose(distance, 3, 2);
        
        image = image.contiguous();
        distance = distance.contiguous();
        geodesic_frontback_pass_cuda(image, distance, {spacing[1], spacing[0], spacing[2]}, l_grad, l_eucl);
        
        // transpose back to original depth, height, width
        image = torch::transpose(image, 3, 2);
        distance = torch::transpose(distance, 3, 2);
        
        // left-right - width*, height, depth
        image = torch::transpose(image, 4, 2);
        distance = torch::transpose(distance, 4, 2);
        
        image = image.contiguous();
        distance = distance.contiguous();
        geodesic_frontback_pass_cuda(image, distance, {spacing[2], spacing[1], spacing[0]}, l_grad, l_eucl);
        
        // transpose back to original depth, height, width
        image = torch::transpose(image, 4, 2);
        distance = torch::transpose(distance, 4, 2);

        // * indicates the current direction of pass
    }

    return distance;
}